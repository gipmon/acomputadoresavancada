#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA
// sgm algorithm adapted from http://lunokhod.org/?p=1403

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>
#include <assert.h>
#include <float.h>
#include <stdlib.h>

#include <limits>
#include <algorithm>

// includes, project
#include <cutil_inline.h>

#define MMAX_BRIGHTNESS 255

#define PENALTY1 15
#define PENALTY2 100
#define NPP_MAX_16U 65535

#define COSTS(i,j,d)              costs[(i)*disp_range+(j)*nx*disp_range+(d)]
#define ACCUMULATED_COSTS(i,j,d)  accumulated_costs[(i)*disp_range+(j)*nx*disp_range+(d)]
#define LEFT_IMAGE(i,j)           left_image[(i)+(j)*nx]
#define RIGHT_IMAGE(i,j)          right_image[(i)+(j)*nx]
#define DISP_IMAGE(i,j)           disp_image[(i)+(j)*nx]

#define MMAX(a,b) (((a)>(b))?(a):(b))
#define MMIN(a,b) (((a)<(b))?(a):(b))

/* function headers */

void determine_costs(const int *left_image, const int *right_image, int *costs,
                     const int nx, const int ny, const int disp_range);

void evaluate_path( const int *prior, const int* local,
                    int path_intensity_gradient, int *curr_cost,
                    const int nx, const int ny, const int disp_range );
__device__ void evaluate_path_dev(const int *prior, const int *local,
                    int path_intensity_gradient, int *curr_cost ,
                    const int nx, const int ny, const int disp_range, const int d, int shmem[]);

void iterate_direction_dirxpos(const int dirx, const int *left_image,
                               const int* costs, int *accumulated_costs,
                               const int nx, const int ny, const int disp_range );

void iterate_direction_dirypos(const int diry, const int *left_image,
                               const int* costs, int *accumulated_costs,
                               const int nx, const int ny, const int disp_range );

void iterate_direction_dirxneg(const int dirx, const int *left_image,
                               const int* costs, int *accumulated_costs,
                               const int nx, const int ny, const int disp_range );

void iterate_direction_diryneg(const int diry, const int *left_image,
                               const int* costs, int *accumulated_costs,
                               const int nx, const int ny, const int disp_range );

void iterate_direction( const int dirx, const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range ) ;

void inplace_sum_views( int * im1, const int * im2,
                        const int nx, const int ny, const int disp_range ) ;

int find_min_index( const int *v, const int dist_range ) ;

void create_disparity_view( const int *accumulated_costs , int * disp_image, int nx, int ny) ;

void sgmHost(   const int *h_leftIm, const int *h_rightIm,
                int *h_dispIm,
                const int w, const int h, const int disp_range );

void sgmDevice( const int *h_leftIm, const int *h_rightIm,
                int *h_dispImD,
                const int w, const int h, const int disp_range );

void usage(char *command);


/* functions code */

void determine_costs(const int *left_image, const int *right_image, int *costs,
                     const int nx, const int ny, const int disp_range)
{
  std::fill(costs, costs+nx*ny*disp_range, 255u);

  for ( int j = 0; j < ny; j++ ) {
      for ( int d = 0; d < disp_range; d++ ) {
          for ( int i = d; i < nx; i++ ) {
              COSTS(i,j,d) = abs( LEFT_IMAGE(i,j) - RIGHT_IMAGE(i-d,j) );
        }
      }
  }
}

__global__ void determine_costs_device(const int *left_image, const int *right_image, int *costs,
                                        const int nx, const int ny, const int disp_range)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < nx && j < ny)
  {
    for ( int d = 0; d < disp_range; d++ ) {
      if(i >= d){
        COSTS(i,j,d) = abs( LEFT_IMAGE(i,j) - RIGHT_IMAGE(i-d,j));
      }
    }
  }
}

void iterate_direction_dirxpos(const int dirx, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    const int WIDTH = nx;
    const int HEIGHT = ny;

      for ( int j = 0; j < HEIGHT; j++ ) {
          for ( int i = 0; i < WIDTH; i++ ) {
              if(i==0) {
                  for ( int d = 0; d < disp_range; d++ ) {
                      ACCUMULATED_COSTS(0,j,d) += COSTS(0,j,d);
                  }
              }
              else {
                  evaluate_path( &ACCUMULATED_COSTS(i-dirx,j,0),
                                 &COSTS(i,j,0),
                                 abs(LEFT_IMAGE(i,j)-LEFT_IMAGE(i-dirx,j)) ,
                                 &ACCUMULATED_COSTS(i,j,0), nx, ny, disp_range);
              }
          }
      }
}

__global__ void iterate_direction_dirxpos_dev(const int dirx, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range ){

      int i = threadIdx.x;
      int j = blockIdx.y * blockDim.y + threadIdx.y;
      extern __shared__ int shmem[];

      if(i < disp_range && j<ny){
        ACCUMULATED_COSTS(0,j,i) += COSTS(0,j,i);
        shmem[i] = COSTS(0,j,i);
      __syncthreads();


      for(int l = 1; l<nx;l++){
        evaluate_path_dev( &ACCUMULATED_COSTS(l-dirx,j,0),
                         &COSTS(l,j,0),
                         abs(LEFT_IMAGE(l,j)-LEFT_IMAGE(l-dirx,j)) ,
                         &ACCUMULATED_COSTS(l,j,0), nx, ny, disp_range, i, shmem);

        __syncthreads();

      }
    }



}

void iterate_direction_dirypos(const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    const int WIDTH = nx;
    const int HEIGHT = ny;

      for ( int i = 0; i < WIDTH; i++ ) {
          for ( int j = 0; j < HEIGHT; j++ ) {
              if(j==0) {
                  for ( int d = 0; d < disp_range; d++ ) {
                      ACCUMULATED_COSTS(i,0,d) += COSTS(i,0,d);
                  }
              }
              else {
                  evaluate_path( &ACCUMULATED_COSTS(i,j-diry,0),
                                 &COSTS(i,j,0),
                                 abs(LEFT_IMAGE(i,j)-LEFT_IMAGE(i,j-diry)),
                                 &ACCUMULATED_COSTS(i,j,0), nx, ny, disp_range );
              }
          }
      }
}

__global__ void iterate_direction_dirypos_dev(const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = threadIdx.y;
    extern __shared__ int shmem[];

    if(j < disp_range && i < nx){
        shmem[j] = COSTS(i,0,j);
        ACCUMULATED_COSTS(i,0,j) += COSTS(i,0,j);
        __syncthreads();

        for(int l = 1; l<ny; l++){

          evaluate_path_dev( &ACCUMULATED_COSTS(i,l-diry,0),
                         &COSTS(i,l,0),
                         abs(LEFT_IMAGE(i,l)-LEFT_IMAGE(i,l-diry)),
                         &ACCUMULATED_COSTS(i,l,0), nx, ny, disp_range, j,shmem);
          __syncthreads();

      }
    }
}

void iterate_direction_dirxneg(const int dirx, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    const int WIDTH = nx;
    const int HEIGHT = ny;

      for ( int j = 0; j < HEIGHT; j++ ) {
          for ( int i = WIDTH-1; i >= 0; i-- ) {
              if(i==WIDTH-1) {
                  for ( int d = 0; d < disp_range; d++ ) {
                      ACCUMULATED_COSTS(WIDTH-1,j,d) += COSTS(WIDTH-1,j,d);
                  }
              }
              else {
                  evaluate_path( &ACCUMULATED_COSTS(i-dirx,j,0),
                                 &COSTS(i,j,0),
                                 abs(LEFT_IMAGE(i,j)-LEFT_IMAGE(i-dirx,j)),
                                 &ACCUMULATED_COSTS(i,j,0), nx, ny, disp_range );
              }
          }
      }
}

__global__ void iterate_direction_dirxneg_dev(const int dirx, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
      int i = threadIdx.x;
      int j = blockIdx.y * blockDim.y + threadIdx.y;
      extern __shared__ int shmem[];

      if(i < disp_range && j < ny){
        shmem[i] = COSTS(nx-1,j,i);

        ACCUMULATED_COSTS(nx-1,j,i) += COSTS(nx-1,j,i);

        __syncthreads();


        for(int l = nx-2; l >= 0; l--){
            evaluate_path_dev( &ACCUMULATED_COSTS(l-dirx,j,0),
                           &COSTS(l,j,0),
                           abs(LEFT_IMAGE(l,j)-LEFT_IMAGE(l-dirx,j)),
                           &ACCUMULATED_COSTS(l,j,0), nx, ny, disp_range, i, shmem);
            __syncthreads();


        }
      }
}

void iterate_direction_diryneg(const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    const int WIDTH = nx;
    const int HEIGHT = ny;

      for ( int i = 0; i < WIDTH; i++ ) {
          for ( int j = HEIGHT-1; j >= 0; j-- ) {
              if(j==HEIGHT-1) {
                  for ( int d = 0; d < disp_range; d++ ) {
                      ACCUMULATED_COSTS(i,HEIGHT-1,d) += COSTS(i,HEIGHT-1,d);
                  }
              }
              else {
                  evaluate_path( &ACCUMULATED_COSTS(i,j-diry,0),
                           &COSTS(i,j,0),
                           abs(LEFT_IMAGE(i,j)-LEFT_IMAGE(i,j-diry)),
                           &ACCUMULATED_COSTS(i,j,0) , nx, ny, disp_range);
             }
         }
      }
}

__global__ void iterate_direction_diryneg_dev(const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{

      int i = blockIdx.x * blockDim.x + threadIdx.x;
      int j = threadIdx.y;
      extern __shared__ int shmem[];

      if(j < disp_range && i < nx){
        shmem[j] = COSTS(i,ny-1,j);

        ACCUMULATED_COSTS(i,ny-1,j) += COSTS(i,ny-1,j);
        __syncthreads();


        for(int l = ny-2; l >= 0; l--){

            evaluate_path_dev( &ACCUMULATED_COSTS(i,l-diry,0),
                       &COSTS(i,l,0),
                       abs(LEFT_IMAGE(i,l)-LEFT_IMAGE(i,l-diry)),
                       &ACCUMULATED_COSTS(i,l,0) , nx, ny, disp_range, j, shmem);
            __syncthreads();

         }
      }
}

void iterate_direction( const int dirx, const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    // Walk along the edges in a clockwise fashion
    if ( dirx > 0 ) {
      // LEFT MOST EDGE
      // Process every pixel along this edge

      iterate_direction_dirxpos(dirx,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
    else if ( diry > 0 ) {
      // TOP MOST EDGE
      // Process every pixel along this edge only if dirx ==
      // 0. Otherwise skip the top left most pixel
      //iterate_direction_dirypos(diry,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
    else if ( dirx < 0 ) {
      // RIGHT MOST EDGE
      // Process every pixel along this edge only if diry ==
      // 0. Otherwise skip the top right most pixel
      //iterate_direction_dirxneg(dirx,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
    else if ( diry < 0 ) {
      // BOTTOM MOST EDGE
      // Process every pixel along this edge only if dirx ==
      // 0. Otherwise skip the bottom left and bottom right pixel
      //iterate_direction_diryneg(diry,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
}
void iterate_direction_dev( const int dirx, const int diry, const int *left_image,
                        const int* costs, int *accumulated_costs,
                        const int nx, const int ny, const int disp_range )
{
    // Walk along the edges in a clockwise fashion
    if ( dirx > 0 ) {
      // LEFT MOST EDGE
      int block_x = disp_range;
      int block_y = 1;

      int grid_x = ceil((float)nx / block_x);
      int grid_y = ceil((float)ny / block_y);

      dim3 block(block_x, block_y);
      dim3 grid(1, grid_y);
      // Process every pixel along this edge
      iterate_direction_dirxpos_dev<<<grid, block, disp_range*sizeof(int)>>>(dirx,left_image,costs,accumulated_costs, nx, ny, disp_range);


    }
    else if ( diry > 0 ) {
      // TOP MOST EDGE
      int block_x = 1;
      int block_y = disp_range;

      int grid_x = ceil((float)nx / block_x);
      int grid_y = ceil((float)ny / block_y);

      dim3 block(block_x, block_y);
      dim3 grid(grid_x, 1);
      // Process every pixel along this edge only if dirx ==
      // 0. Otherwise skip the top left most pixel
      //iterate_direction_dirypos_dev<<<grid, block, disp_range*sizeof(int)>>>(diry,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
    else if ( dirx < 0 ) {
      // RIGHT MOST EDGE
      int block_x = disp_range;
      int block_y = 1;

      int grid_x = ceil((float)nx / block_x);
      int grid_y = ceil((float)ny / block_y);

      dim3 block(block_x, block_y);
      dim3 grid(1, grid_y);
      // Process every pixel along this edge only if diry ==
      // 0. Otherwise skip the top right most pixel
      //iterate_direction_dirxneg_dev<<<grid, block, disp_range*sizeof(int)>>>(dirx,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
    else if ( diry < 0 ) {
      // BOTTOM MOST EDGE
      int block_x = 1;
      int block_y = disp_range;

      int grid_x = ceil((float)nx / block_x);
      int grid_y = ceil((float)ny / block_y);

      dim3 block(block_x, block_y);
      dim3 grid(grid_x, 1);
      // Process every pixel along this edge only if dirx ==
      // 0. Otherwise skip the bottom left and bottom right pixel
      //iterate_direction_diryneg_dev<<<grid, block, disp_range*sizeof(int)>>>(diry,left_image,costs,accumulated_costs, nx, ny, disp_range);
    }
}

// ADD two cost images
void inplace_sum_views( int * im1, const int * im2,
                        const int nx, const int ny, const int disp_range )
{
    int *im1_init = im1;
    while ( im1 != (im1_init + (nx*ny*disp_range)) ) {
      *im1 += *im2;
      im1++;
      im2++;
    }
}

__global__ void inplace_sum_views_dev(int * im1, const int * im2,
                                      const int nx, const int ny, const int disp_range){
      int i = blockIdx.x * blockDim.x + threadIdx.x;
      int j = blockIdx.y * blockDim.y + threadIdx.y;
      int id = i + (j * (nx*disp_range));
      if(i < nx*disp_range && j < ny){
        int *im1_init = im1;
        im1 += id;
        im2 += id;
        if(im1 != (im1_init + (nx*ny*disp_range))  ){
          *im1 += *im2;
        }
      }

}

int find_min_index( const int *v, const int disp_range )
{
    int min = std::numeric_limits<int>::max();
    int minind = -1;
    for (int d=0; d < disp_range; d++) {
         if(v[d]<min) {
              min = v[d];
              minind = d;
         }
    }
    return minind;
}

__device__ int find_min_index_device( const int *v, const int disp_range )
{
    int min = NPP_MAX_16U;
    int minind = -1;
    for (int d=0; d < disp_range; d++) {
         if(v[d]<min) {
              min = v[d];
              minind = d;
         }
    }
    return minind;
}

void evaluate_path(const int *prior, const int *local,
                   int path_intensity_gradient, int *curr_cost ,
                   const int nx, const int ny, const int disp_range)
{
  memcpy(curr_cost, local, sizeof(int)*disp_range);

  for ( int d = 0; d < disp_range; d++ ) {
    int e_smooth = std::numeric_limits<int>::max();
    for ( int d_p = 0; d_p < disp_range; d_p++ ) {
      if ( d_p - d == 0 ) {
        // No penality
        e_smooth = MMIN(e_smooth,prior[d_p]);
      } else if ( abs(d_p - d) == 1 ) {
        // Small penality
        e_smooth = MMIN(e_smooth,prior[d_p]+PENALTY1);
      } else {
        // Large penality
        e_smooth =
          MMIN(e_smooth,prior[d_p] +
                   MMAX(PENALTY1,
                            path_intensity_gradient ? PENALTY2/path_intensity_gradient : PENALTY2));
      }
    }
    curr_cost[d] += e_smooth;
  }

  int min = std::numeric_limits<int>::max();
  for ( int d = 0; d < disp_range; d++ ) {
        if (prior[d]<min) min=prior[d];
  }
  for ( int d = 0; d < disp_range; d++ ) {
        curr_cost[d]-=min;
  }
}

__device__ void evaluate_path_dev(const int *prior, const int *local,
                     int path_intensity_gradient, int *curr_cost ,
                     const int nx, const int ny, const int disp_range, const int d, int shmem[])
  {
    memcpy(curr_cost, local, sizeof(int)*disp_range);
    int e_smooth = NPP_MAX_16U;

    for ( int d_p = 0; d_p < disp_range; d_p++ ) {
      if ( d_p - d == 0 ) {
        // No penality
        e_smooth = MMIN(e_smooth,shmem[d_p]);
      } else if ( abs(d_p - d) == 1 ) {
        // Small penality
        e_smooth = MMIN(e_smooth,shmem[d_p]+PENALTY1);
      } else {
        // Large penality
        e_smooth =
          MMIN(e_smooth,shmem[d_p] +
                   MMAX(PENALTY1,
                            path_intensity_gradient ? PENALTY2/path_intensity_gradient : PENALTY2));
      }
    }

    curr_cost[d] += e_smooth;

    int min = NPP_MAX_16U;
    for ( int d_s = 0; d_s < disp_range; d_s++ ) {
      if (shmem[d_s]<min) min=shmem[d_s];
    }
    curr_cost[d]-=min;
    __syncthreads();
    shmem[d] = curr_cost[d];

}

void create_disparity_view( const int *accumulated_costs , int * disp_image,
                            const int nx, const int ny, const int disp_range)
{
  for ( int j = 0; j < ny; j++ ) {
    for ( int i = 0; i < nx; i++ ) {
      DISP_IMAGE(i,j) =
        4 * find_min_index( &ACCUMULATED_COSTS(i,j,0), disp_range );
    }
  }
}

__global__ void create_disparity_view_dev(int *disp_image, int *accumulated_costs, int nx, int ny, int disp_range){
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;

  if (i < nx && j < ny)
  {
    DISP_IMAGE(i,j) = 4 * find_min_index_device(&ACCUMULATED_COSTS(i,j,0), disp_range);
  }

}

/*
 * Links:
 * http://www.dlr.de/rmc/rm/en/desktopdefault.aspx/tabid-9389/16104_read-39811/
 * http://lunokhod.org/?p=1356
 */

// sgm code to run on the host
void sgmHost(   const int *h_leftIm, const int *h_rightIm,
                int *h_dispIm,
                const int w, const int h, const int disp_range)
{
    const int nx = w;
    const int ny = h;

  // Processing all costs. W*H*D. D= disp_range
  int *costs = (int *) calloc(nx*ny*disp_range,sizeof(int));
  if (costs == NULL) {
        fprintf(stderr, "sgm_cuda:"
                " Failed memory allocation(s).\n");
        exit(1);
  }

  determine_costs(h_leftIm, h_rightIm, costs, nx, ny, disp_range);

  int *accumulated_costs = (int *) calloc(nx*ny*disp_range,sizeof(int));
  int *dir_accumulated_costs = (int *) calloc(nx*ny*disp_range,sizeof(int));
  if (accumulated_costs == NULL || dir_accumulated_costs == NULL) {
        fprintf(stderr, "sgm_cuda:"
                " Failed memory allocation(s).\n");
        exit(1);
  }

  int dirx=0,diry=0;
  for(dirx=-1; dirx<2; dirx++) {
      if(dirx==0 && diry==0) continue;
      std::fill(dir_accumulated_costs, dir_accumulated_costs+nx*ny*disp_range, 0);
      iterate_direction( dirx,diry, h_leftIm, costs, dir_accumulated_costs, nx, ny, disp_range);
      inplace_sum_views( accumulated_costs, dir_accumulated_costs, nx, ny, disp_range);
  }
  dirx=0;
  for(diry=-1; diry<2; diry++) {
      if(dirx==0 && diry==0) continue;
      std::fill(dir_accumulated_costs, dir_accumulated_costs+nx*ny*disp_range, 0);
      iterate_direction( dirx,diry, h_leftIm, costs, dir_accumulated_costs, nx, ny, disp_range);
      inplace_sum_views( accumulated_costs, dir_accumulated_costs, nx, ny, disp_range);
  }

  free(costs);
  free(dir_accumulated_costs);

  create_disparity_view( accumulated_costs, h_dispIm, nx, ny, disp_range );

  free(accumulated_costs);
}


// sgm code to run on the GPU
void sgmDevice( const int *h_leftIm, const int *h_rightIm,
                int *h_dispImD,
                const int w, const int h, const int disp_range )
{
  const int nx = w;
  const int ny = h;
  int imageSize = nx * ny * sizeof(int);

  int block_x = 32;
  int block_y = 16;

  int grid_x = ceil((float)nx / block_x);
  int grid_y = ceil((float)ny / block_y);

  int grid1_x = ceil((float)nx*disp_range / block_x);

  dim3 block(block_x, block_y);
  dim3 grid(grid_x, grid_y);
  dim3 grid1(grid1_x, grid_y);

  // Processing all costs. W*H*D. D= disp_range
  int *costs = (int *) calloc(nx*ny*disp_range,sizeof(int));
  if (costs == NULL) {
        fprintf(stderr, "sgm_cuda:"
                " Failed memory allocation(s).\n");
        exit(1);
  }

  int *devPtr_leftImage;
  int *devPtr_rightImage;
  int *devPtr_costs;
  int *devPtr_dirAccumulatedCosts;
  int *devPtr_accumulatedCosts;
  int *devPtr_hDispImD;

  std::fill(costs, costs+nx*ny*disp_range, 255u);

  hipMalloc((void**)&devPtr_leftImage, imageSize);
  hipMalloc((void**)&devPtr_rightImage, imageSize);
  hipMalloc((void**)&devPtr_hDispImD, imageSize);
  hipMalloc((void**)&devPtr_costs, nx*ny*disp_range*sizeof(int));

  hipMemcpy(devPtr_leftImage, h_leftIm, imageSize, hipMemcpyHostToDevice);
  hipMemcpy(devPtr_rightImage, h_rightIm, imageSize, hipMemcpyHostToDevice);
  hipMemcpy(devPtr_costs, costs, nx*ny*disp_range*sizeof(int), hipMemcpyHostToDevice);

  determine_costs_device<<<grid, block>>>(devPtr_leftImage, devPtr_rightImage, devPtr_costs, nx, ny, disp_range);

  hipMalloc((void**)&devPtr_dirAccumulatedCosts, nx*ny*disp_range*sizeof(int));
  hipMalloc((void**)&devPtr_accumulatedCosts, nx*ny*disp_range*sizeof(int));

  hipMemset(devPtr_accumulatedCosts, 0, nx*ny*disp_range*sizeof(int));


  int dirx=0,diry=0;
  for(dirx=-1; dirx<2; dirx++) {
      if(dirx==0 && diry==0) continue;
      hipMemset(devPtr_dirAccumulatedCosts, 0, nx*ny*disp_range*sizeof(int));
      iterate_direction_dev( dirx,diry, devPtr_leftImage, devPtr_costs, devPtr_dirAccumulatedCosts, nx, ny, disp_range);
      inplace_sum_views_dev<<<grid1, block>>>( devPtr_accumulatedCosts, devPtr_dirAccumulatedCosts, nx, ny, disp_range);

  }
  dirx=0;
  for(diry=-1; diry<2; diry++) {
      if(dirx==0 && diry==0) continue;
      hipMemset(devPtr_dirAccumulatedCosts, 0, nx*ny*disp_range*sizeof(int));
      iterate_direction_dev( dirx,diry, devPtr_leftImage, devPtr_costs, devPtr_dirAccumulatedCosts, nx, ny, disp_range);
      inplace_sum_views_dev<<<grid1, block>>>( devPtr_accumulatedCosts, devPtr_dirAccumulatedCosts, nx, ny, disp_range);
  }

  create_disparity_view_dev<<<grid, block>>> (devPtr_hDispImD, devPtr_accumulatedCosts, nx, ny, disp_range);

  hipMemcpy(h_dispImD, devPtr_hDispImD, imageSize, hipMemcpyDeviceToHost);

  hipFree(devPtr_leftImage);
  hipFree(devPtr_rightImage);
  hipFree(devPtr_hDispImD);
  hipFree(devPtr_costs);
  hipFree(devPtr_dirAccumulatedCosts);
  hipFree(devPtr_accumulatedCosts);
}

// print command line format
void usage(char *command)
{
    printf("Usage: %s [-h] [-d device] [-l leftimage] [-r rightimage] [-o dev_dispimage] [-t host_dispimage] [-p disprange] \n",command);
}

// main
int main( int argc, char** argv)
{

    // default command line options
    int deviceId = 0;
    int disp_range = 32;
    char *leftIn      =(char *)"lbull.pgm",
         *rightIn     =(char *)"rbull.pgm",
         *fileOut     =(char *)"d_dbull.pgm",
         *referenceOut=(char *)"h_dbull.pgm";

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:l:o:r:t:p:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':  // device
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'l': // left image filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                leftIn = strdup(optarg);
                break;
            case 'r': // right image filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                rightIn = strdup(optarg);
                break;
            case 'o': // output image (from device) filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 't': // output image (from host) filename
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'p': // disp_range
                if(sscanf(optarg,"%d",&disp_range)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'h': // help
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    if(optind < argc) {
        fprintf(stderr,"Error in arguments\n");
        usage(argv[0]);
        exit(1);
    }

    // select cuda device
    cutilSafeCall( hipSetDevice( deviceId ) );

    // create events to measure host sgm time and device sgm time
    hipEvent_t startH, stopH, startD, stopD;
    hipEventCreate(&startH);
    hipEventCreate(&stopH);
    hipEventCreate(&startD);
    hipEventCreate(&stopD);

    // allocate host memory
    int* h_ldata=NULL;
    int* h_rdata=NULL;
    unsigned int h,w;

    //load left pgm
    if (cutLoadPGMi(leftIn, (unsigned int **)&h_ldata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", leftIn);
        exit(1);
    }
    //load right pgm
    if (cutLoadPGMi(rightIn, (unsigned int **)&h_rdata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", rightIn);
        exit(1);
    }

    // allocate mem for the result on host side
    int* h_odata = (int*) malloc( h*w*sizeof(int));
    int* reference = (int*) malloc( h*w*sizeof(int));

    // sgm at host
    hipEventRecord( startH, 0 );
    sgmHost(h_ldata, h_rdata, reference, w, h, disp_range);
    hipEventRecord( stopH, 0 );
    hipEventSynchronize( stopH );

    // sgm at GPU
    hipEventRecord( startD, 0 );
    sgmDevice(h_ldata, h_rdata, h_odata, w, h, disp_range);
    hipEventRecord( stopD, 0 );
    hipEventSynchronize( stopD );

    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    float timeH, timeD;
    hipEventElapsedTime( &timeH, startH, stopH );
    printf( "Host processing time: %f (ms)\n", timeH);
    hipEventElapsedTime( &timeD, startD, stopD );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (cutSavePGMi(referenceOut, (unsigned int *)reference, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (cutSavePGMi(fileOut,(unsigned int *) h_odata, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    cutFree( h_ldata);
    cutFree( h_rdata);
    free( h_odata);
    free( reference);

    cutilDeviceReset();
}
