
#include <hip/hip_runtime.h>
#include  <stdio.h>
#include  <time.h>

#define  SIZE (16)

//  Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide
__global__  void arrAdd(float* A, float* B, float* C)
{
    // TODO: determine id
	int id;

	id = blockIdx.x * blockDim.x + threadIdx.x;

    C[id] = A[id] + B[id];
}

int  main(void)
{
	float A[SIZE], B[SIZE], C[SIZE];
	float *devPtrA;
	float *devPtrB;
	float *devPtrC;
	int memsize = SIZE * sizeof(float);

	// Initialize arrays
	srand (time(NULL));
	for(int i=0; i < SIZE; i++)
	{
		A[i]=rand() % 100;
		B[i]=rand() % 100;
	}

	hipSetDevice(0);	// Select GPU device (can be 0 to 3)

	// Allocate device memory for A, B and C arrays
	hipMalloc((void**)&devPtrA, memsize);
	hipMalloc((void**)&devPtrB, memsize);
	hipMalloc((void**)&devPtrC, memsize);

	// Copy data (data to process) from host to device (from CPU to GPU)
	hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice);
	hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice);

	// Execute the Kernell
	// TO DO: Determine the number of blocks
	arrAdd <<<8, 2>>> (devPtrA,  devPtrB, devPtrC); // launch ??? blocks with 2 threads each

	// Copy data from device (results) back to host
	hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost);

	// Show results
	printf("     A      B       C\n");
	for (int i=0; i < SIZE; i++)
	{
		printf("%2d: %4.1f + %4.1f = %5.1f\n", i, A[i], B[i], C[i]);
	}

	hipFree(devPtrA);
	hipFree(devPtrB);
	hipFree(devPtrC);
}
