#include "hip/hip_runtime.h"
#include  <stdio.h> 
#include  <time.h> 

#define  SIZE (16) 

//  Kernel definition, see also section 4.2.3 of Nvidia Cuda Programming Guide 
__global__  void arrAdd(float* A, float* B, float* C) 
{ 
	// TODO: determine id
	int id;

	id = ...

	if(id < SIZE)
	{
		C[id] = A[id] + B[id]; 
	}
} 
 
int  main(void) 
{ 
	float A[SIZE], B[SIZE], C[SIZE]; 
	float *devPtrA; 
	float *devPtrB; 
	float *devPtrC; 
	int memsize = SIZE * sizeof(float); 

	// Initialize arrays
	srand (time(NULL));
	for(int i=0; i < SIZE; i++) 
	{
		A[i]=rand() % 100;
		B[i]=rand() % 100;
	}

	hipSetDevice(0);	// Select GPU device (can be 0 to 3)

	// Allocate device memory for A, B and C arrays
	hipMalloc((void**)&devPtrA, memsize); 
	hipMalloc((void**)&devPtrB, memsize); 
	hipMalloc((void**)&devPtrC, memsize); 

	// Copy data (data to process) from host to device (from CPU to GPU)
	hipMemcpy(devPtrA, A, memsize,  hipMemcpyHostToDevice); 
	hipMemcpy(devPtrB, B, memsize,  hipMemcpyHostToDevice); 

	// __global__ functions are called:  Func <<< dim grid, dim block >>> (parameter); 
	dim3 dimGrid( /*TODO*/);
	dim3 dimBlock(/*TODO*/);

	// Execute the Kernell
	arrAdd <<<dimGrid, dimBlock>>> (devPtrA,  devPtrB, devPtrC); 

	// Copy data from device (results) back to host 
	hipMemcpy(C, devPtrC, memsize,  hipMemcpyDeviceToHost); 

	// Show results
	printf("     A      B       C\n");
	for (int i=0; i < SIZE; i++) 
	{
		printf("%2d: %4.1f + %4.1f = %5.1f\n", i, A[i], B[i], C[i]); 
	}

	hipFree(devPtrA); 
	hipFree(devPtrB); 
	hipFree(devPtrC); 
} 


